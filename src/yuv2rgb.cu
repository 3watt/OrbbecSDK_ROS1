#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "orbbec_camera/yuv2rgb.h"

uint8_t *d_y = NULL, *d_u = NULL, *d_v = NULL, *d_rgb = NULL;

extern "C" void yuv422InitMemory(int width, int height) {
  hipMalloc((void**)&d_y, width * height);
  hipMalloc((void**)&d_u, width * height / 4);
  hipMalloc((void**)&d_v, width * height / 4);
  hipMalloc((void**)&d_rgb, width * height * 3);
}

extern "C" void yuv422FreeMemory() {
  hipFree(d_y);
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_rgb);
}

__global__ void yuv_to_rgb_kernel(uint8_t* y_plane, uint8_t* u_plane, uint8_t* v_plane,
                                  uint8_t* rgb, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    float y_val = y_plane[y * width + x];
    float v_val = u_plane[(y / 2) * (width / 2) + (x / 2)] - 128;
    float u_val = v_plane[(y / 2) * (width / 2) + (x / 2)] - 128;

    float r = y_val + 1.402 * v_val;
    float g = y_val - 0.344 * u_val - 0.714 * v_val;
    float b = y_val + 1.772 * u_val;
    // printf("r: %f, g: %f, b: %f\n", r, g, b);

    rgb[(y * width + x) * 3 + 0] = r > 255.0 ? 255 : (r < 0.0 ? 0 : static_cast<uint8_t>(r));
    rgb[(y * width + x) * 3 + 1] = g > 255.0 ? 255 : (g < 0.0 ? 0 : static_cast<uint8_t>(g));
    rgb[(y * width + x) * 3 + 2] = b > 255.0 ? 255 : (b < 0.0 ? 0 : static_cast<uint8_t>(b));
  }
}

extern "C" void yuv422ToRgb(uint8_t* y_plane, uint8_t* u_plane, uint8_t* v_plane,
                            unsigned char* rgb, int width, int height) {
  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
  hipMemcpy(d_y, y_plane, width * height, hipMemcpyHostToDevice);
  hipMemcpy(d_u, u_plane, width * height / 4, hipMemcpyHostToDevice);
  hipMemcpy(d_v, v_plane, width * height / 4, hipMemcpyHostToDevice);
  yuv_to_rgb_kernel<<<grid, block>>>(d_y, d_u, d_v, d_rgb, width, height);
  // get cuda err
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }
  hipMemcpy(rgb, d_rgb, width * height * 3, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
}
